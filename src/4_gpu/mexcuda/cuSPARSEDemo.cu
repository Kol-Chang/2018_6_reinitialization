/*==============================================================================
 *Filename: cuSPARSEDemo.cu
 * Description: This function implements a sparse matrix-vector
 * multiplication (MEX file that contains CUDA code and takes
 * as inputs MATLAB arrays)
 * Authors: Ploskas, N., & Samaras, N.
 * Syntax: x = cuSPARSEDemo(A, b)
 * Input:
 * -- A: a sparse double-precision, floating point array of size NxN
 * -- b: a double-precision, floating point vector of size Nx1
 * Output:
 * -- x: a double-precision, floating point vector of size Nx1
 *============================================================================*/

#include "mex.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>

/*
* The gateway function
*/
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){

	/* declare all variables */
	double *s, *b, *x;
	double *d_s, *d_b, *d_x;
	mwSize *iir, *jjc;
	int *ir, *jc;
	int *d_ir, *d_jc;
	int N, nz;

	/* define error messages */
	char const * const errId = "parallel:gpu:cuSPARSEDemo:InvalidInput";
	char const * const errMsg = "Invalid input to MEX file.";

	/* check input data */
	if(nrhs != 2 || !mxIsSparse(prhs[0]) || !mxIsDouble(prhs[0]) || !mxIsDouble(prhs[0]) || mxGetM(prhs[0]) != mxGetN(prhs[0]) || mxGetN(prhs[0]) != mxGetM(prhs[1])){
		mexErrMsgIdAndTxt(errId, errMsg);
	}

	/* initialize output array */
	plhs[0] = mxCreateDoubleMatrix(mxGetM(prhs[0]), 1, mxREAL);

	/* get input arrays */
	iir = mxGetIr(prhs[0]); /* Row indexing */
	jjc = mxGetJc(prhs[0]); /* Column count */
	s = mxGetPr(prhs[0]); /* Nonzero elements */
	b = mxGetPr(prhs[1]); /* Rhs vector */
	x = mxGetPr(plhs[0]); /* Output vector */

	/* find arrays dimensions */
	N = mxGetN(prhs[0]);

	/* find the number of nonzero elements */
	nz = (int) mxGetNzmax(prhs[0]);

	/* convert row indexing and column count from mwsize 
	   to int */
	ir = new int[nz];
	jc = new int[N + 1];
	for(int i = 0; i < nz; i++){
		ir[i] = (int)(iir[i]);
	}
	for(int i = 0; i <= N; i++){
	jc[i] = (int)(jjc[i]);
	}

	/* allocate memory on the GPU */
	hipMalloc((void **)&d_ir, nz * sizeof(int));
	hipMalloc((void **)&d_jc, (N + 1) * sizeof(int));
	hipMalloc((void **)&d_s, nz * sizeof(double));
	hipMalloc((void **)&d_x, N * sizeof(double));
	hipMalloc((void **)&d_b, N * sizeof(double));
	hipMemcpy(d_ir, ir, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_jc, jc, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_s, s, nz * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);

	/* create handle */
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseCreate(&cusparseHandle);

	/* create and set matrix description and matrix
	   index base */
	hipsparseMatDescr_t descr = 0;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	/* perform the sparse matrix-vector multiplication
	   using cuSPARSE */
	double alpha = 1.0;
	double beta = 0.0;
	hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, N, N, nz, &alpha, descr, d_s, d_jc, d_ir, d_b, &beta, d_x);

	/* copy result */
	hipMemcpy(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost);

	/* destroy the handle and the arrays on the device */
	hipsparseDestroy(cusparseHandle);
	hipFree(d_s);
	hipFree(d_ir);
	hipFree(d_jc);
	hipFree(d_b);
	hipFree(d_x);
}
