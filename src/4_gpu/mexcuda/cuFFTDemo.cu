/*==============================================================================
 *Filenmae: cuFFTDemo.cu
 * Description: This function implements a two-dimensional
 * discrete Fourier transform of a two-dimensional array
 * using cuFFT library (MEX file that contains CUDA code and
 * takes as inputs MATLAB arrays)
 * Authors: Ploskas, N., & Samaras, N.
 * Syntax: B = cuFFTDemo(A)
 * Input:
 *   -- A: a double-precision, floating point array of size MxN
 * Output:
 *   -- C: a double-precision, floating point array of size 
 *      (M / 2 + 1)/xN
 *============================================================================*/

#include "mex.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

/*
 * The gateway function
 */
void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
	/* declare all variables */
	double * deviceA;
	hipfftDoubleComplex * deviceB;
	double *A, *B;
	int numARows, numACols;
	int numBRows, numBCols;

	/* define error messages */
	char const * const errId = "parallel:gpu:mexMatrixMultiplication:InvalidInput";
	char const * const errMsg = "Invalid input to MEX file.";
	

	/* check input datea */
	if(nrhs != 1){
		mexErrMsgIdAndTxt(errId, errMsg);
	}

	/* get input array */
	A = (double *)mxGetData(prhs[0]);

	/* find array dimensions */
	numARows = (int)mxGetM(prhs[0]);
	numACols = (int)mxGetN(prhs[0]);

	/* initialize output array */
	numBRows = numARows / 2 + 1;
	numBCols = numACols;

	plhs[0] = mxCreateNumericMatrix(numBRows, numBCols, mxDOUBLE_CLASS, mxREAL);
	B = (double *)mxMalloc(sizeof(hipfftDoubleComplex) * numBRows * numBCols);


	/* allocate memory on the GPU */
	hipMalloc(&deviceA, sizeof(double) * numARows * numACols);
	hipMalloc(&deviceB, sizeof(hipfftDoubleComplex) * numBRows * numBCols);
	hipMemcpy(deviceA, A, numARows * numACols * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(B, deviceB, numBRows * numBCols * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);

	/* create handle and perform the two-dimensional discrete Fourier transform using cuFFT */
	hipfftHandle plan;
	hipfftPlan2d(&plan, numACols, numARows, HIPFFT_D2Z);
	hipfftExecD2Z(plan, deviceA, deviceB);

//	hipMemcpy(B, deviceB, numBRows * numBCols * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);

	/* retrieve results */
	double * real = (double *)mxGetPr(plhs[0]);
	double * imag = (double *)mxGetPi(plhs[0]);
	double * complex_ptr = B;
	for(int i = 0; i < numBCols; ++i){
		for(int j = 0; j < numBRows; ++j){
			*real++ = *complex_ptr++;
			*imag++ = *complex_ptr++;
		}
	}

	/* destroy the handle and the arrays on the device */
	hipfftDestroy(plan);
	hipFree(deviceA);
	hipFree(deviceB);
}
