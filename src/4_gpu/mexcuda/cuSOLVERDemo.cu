/*==============================================================================
 *Filename: cuSOLVERDemo.cu
 *Description: This function solves a system of linear
 *equations Ax = b for x (MEX file that contains CUDA code
 *and takes as inputs MATLAB arrays)
 *Authors: Ploskas, N., & Samaras, N.
 *Syntax: x = cuSOLVERDemo(A, b)
 *Input:
 *  -- A: a double-precision, floating point array of size NxN
 *  -- b: a double-precision, floating point vector of size Nx1
 * Output:
 *	-- x: a double-precision, floating point vector of size Nx1
 *============================================================================*/

#include "mex.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

/*
 * The gateway function
 */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
	/* declare all variables */
	double *deviceA, *deviceB, *deviceX;
	double *A, *b, *x;
	int N;
	int lwork = 0;
	double *d_work = NULL;
	int *devInfo = NULL;
	const double one = 1;

	/* define error messages */
	char const * const errId = "parallel:gpu:cuSOLVERDemo:InvalidInput";
	char const * const errMsg = "Invalid input to MEX file.";

	/* check input data */
	if(nrhs != 2 || !mxIsDouble(prhs[0]) || !mxIsDouble(prhs[1]) || mxGetM(prhs[0]) != mxGetN(prhs[0]) || mxGetM(prhs[0]) != mxGetM(prhs[1])){
		mexErrMsgIdAndTxt(errId, errMsg);
	}

	/* get input arrays */
	A = (double *)mxGetData(prhs[0]);
	b = (double *)mxGetData(prhs[1]);
	
	/* find arrays dimensions */
	N = (int)mxGetN(prhs[0]);
	
	/* initialize output array */
	plhs[0] = mxCreateNumericMatrix(N, 1, mxDOUBLE_CLASS, mxREAL);
	x = (double *)mxGetData(plhs[0]);

	/* allocate memory on the GPU */
	hipMalloc(&deviceA, sizeof(double) * N * N);
	hipMalloc(&deviceB, sizeof(double) * N);
	hipMalloc(&deviceX, sizeof(double) * N);
	hipMemcpy(deviceA, A, sizeof(double) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(deviceB, b, sizeof(double) * N, hipMemcpyHostToDevice);
	
	/* create handle */
	hipsolverHandle_t cudenseH = NULL;
	hipsolverDnCreate(&cudenseH);

	/* query working space of geqrf */
	hipsolverDnDgeqrf_bufferSize(cudenseH, N, N, deviceA, N, &lwork);
	hipMalloc((void**)&d_work, sizeof(double) * lwork);
	hipMalloc((void **)&devInfo, sizeof(int));

	/* compute QR factorization */ 
	hipsolverDnDgeqrf(cudenseH, N, N, deviceA, N, deviceX, d_work, lwork, devInfo);

	/* synchronize the device */
	hipDeviceSynchronize();

	/* compute Q^T*b */
	hipsolverDnDormqr(cudenseH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, N, 1, N, deviceA, N, deviceX, deviceB, N, d_work, lwork, devInfo);

	/* synchronize the device */
	hipDeviceSynchronize();

	/* create handle */
	hipblasHandle_t cublasH = NULL;
	hipblasCreate(&cublasH);

	/* compute x = R \ Q^T*b */
	hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, N, 1, &one, deviceA, N, deviceB, N);

	/* synchronize the device */
	hipDeviceSynchronize();

	/* copy the result */
	hipMemcpy(x, deviceB, sizeof(double) * N, hipMemcpyDeviceToHost);

	/* destroy the handles and the arrays on the device */
	hipblasDestroy(cublasH);
	hipsolverDnDestroy(cudenseH);
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceX);
}
